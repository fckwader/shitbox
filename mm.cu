
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>


void init(float *a, float *b, float *c, int n){
    for(int i = 0; i < n; i++){
        for(int j = 0; j<n; j++){
            a[i*n + j] = 0;
            if(i == j){
                b[i*n +j] = 2;
            }else{
                b[i*n + j] = 0;
            }
            c[i*n + j] = (i* j + 1) % 7;
        }
    }
}

void mm_single(float *a, float *b, float *c, int n){
    for (int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            for(int k = 0; k < n; k++){
                a[i*n + j] += b[i*n + k] * c[k*n + j];
            }
        }
    }
}

__global__
void mm(float *a, float *b, float *c, int n){
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    for(int i = tx; i < n; i+=blockDim.x)
    {
        for(int j = ty; j < n; j+=blockDim.y){
            for(int k = 0; k < n; k++){
                a[i * n + j] += b[i * n + k] * c[k * n + j];
            }
        }
    }
}

void printm(float *a, int n){
printf("-----\n");
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            printf("%.0f ", a[i*n + j]);
        }
        printf("\n");
    }
printf("------\n");
}

int main(){
    int n = 2048;
    dim3 vec(32, 32, 1);
    float *a, *b, *c;
    hipMallocManaged(&a, n*n*sizeof(float));
    hipMallocManaged(&b, n*n*sizeof(float));
    hipMallocManaged(&c, n*n*sizeof(float));

    init(a, b, c, n);
    clock_t start = clock();
        mm<<<1, vec>>>(a, b, c, n);
        hipDeviceSynchronize();
    clock_t end = clock();
    double time_spent = (double)(end - start) / CLOCKS_PER_SEC;
    printf("MM parallel took %.1f seconds\n", time_spent);

    init(a, b, c, n);
    start = clock();
        mm_single(a, b, c, n);
        hipDeviceSynchronize();
    end = clock();
    time_spent = (double)(end - start) / CLOCKS_PER_SEC;
    printf("MM single took %.1f seconds\n", time_spent);


    return 0;

}


