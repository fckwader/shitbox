
#include <hip/hip_runtime.h>
#include <stdio.h>


void init(float *a, float *b, float *c, int n){
    for(int i = 0; i < n; i++){
        for(int j = 0; j<n; j++){
            a[i*n + j] = 0;
            if(i == j){
                b[i*n +j] = 2;
            }else{
                b[i*n + j] = 0;
            }
            c[i*n + j] = (i*j + 1)% 13;
        }
    }
}


__global__
void mm(float *a, float *b, float *c, int n){
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    for(int j = ty; j < n; j+=blockDim.y){
        for(int k = 0; k < n; k++){
            a[tx * n + j] += b[tx * n + k] * c[k * n + j];
        }
    }
}

void printm(float *a, int n){
    for(int i = 0; i < n; i++){
        for(int j = 0; j < n; j++){
            printf("%d ", a[i*n + j]);
        }
        printf("\n");
    }
}

int main(){
    int n = 16;
    dim3 vec(8, 8, 1);
    float a[n*n], b[n*n], c[n*n];

    init(a, b, c, n);
    printm(c, n);
    mm<<<1, vec>>>(a, b, c, n);
    hipDeviceSynchronize();
    printm(a, n);
    return 0;

}


