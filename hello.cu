
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>




__global__
void add(int n, float *x, float *y, float *z)
{

    int blockNr = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;

    int index = blockNr * blockDim.x * blockDim.y * blockDim.z + threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;

    int stride = blockDim.x * blockDim.y * blockDim.z * gridDim.x;

    for (int i = index; i < n; i += stride){
        if(index == 0){
         //printf("0 0 is running i=%d\n", i);
        }
        z[i] = x[i] + y[i];
    }
}

void runBench(int n, float *x, float *y, float *z, int bx, int by, int bz, int dimX, int dimY, int dimZ){
        dim3 blockvec(bx, by, bz);
        dim3 sizevec(dimX, dimY, dimZ);
        //printf("Calc... ");
        clock_t begin = clock();
        add<<<blockvec, sizevec>>>(n, x, y, z);
        hipDeviceSynchronize();
        //printf("Calc complete.\n");
        clock_t end = clock();
        double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
        printf("| %d | %d | %d | %d | %d | %d | %f |\n", bx, by, bz, dimX, dimY, dimZ, time_spent);

        //printf("Verifying... ");
        for(int i = 0; i < n; i++){
            if(z[i] != x[i] + y[i]){
                 printf("ERROR: Expected %d, got %d\n", x[i]+y[i], z[i]);
                 return;
            }
        }
        //printf("Verified. ");

        //printf("Resetting z... ");
        for(int i = 0; i < n; i++){
            z[i] = 0;
        }
        //printf("z reset.\n");
}

int main()
{
    int n = 1000000000;
    float *x, *y, *z;
    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&y, n*sizeof(float));
    hipMallocManaged(&z, n*sizeof(float));

    //init
    printf("Init... ");
    for(int i = 0; i < n; i++){
        x[i] = i % 8;
        y[i] = i % 25;
        z[i] = 0;
    }
    printf("Init complete.\n");

    printf("| BX | BY | BZ | TX | TY | TZ | t |\n");

    runBench(n, x, y, z, 2, 8, 4, 8, 8, 8);
    runBench(n, x, y, z, 4, 4, 4, 8, 8, 8);
    printf("\n");

    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}
