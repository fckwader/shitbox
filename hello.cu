#include "hip/hip_runtime.h"
#include <stdio.h>




__global__
void add(int n, float *x, float *y, float *z)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    if(index == 0){
        printf("Index 0 calculating for i=%d\n", i);
    }
    for(int i = index; i < n; i += stride){
        z[i] = x[i] + y[i];
    }
}

int main()
{
    int n = 1000000;
    float *x, *y, *z;
    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&y, n*sizeof(float));
    hipMallocManaged(&z, n*sizeof(float));

    //init
    printf("Init...\n");
    for(int i = 0; i < n; i++){
        x[i] = i % 8;
        y[i] = i % 25;
        z[i] = 0;
    }
    printf("Init complete\n");

    printf("Calc...\n");
    add<<<1, 256>>>(n, x, y, z);
    hipDeviceSynchronize();
    printf("Calc complete\n");

    for(int i = 0; i < n; i++){
    if(z[i] != x[i] + y[i]){
        printf("ERROR: Expected %d, got %d\n", x[i]+y[i], z[i]);
    }
    }
    printf("\n");

    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}
