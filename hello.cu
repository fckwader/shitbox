
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>




__global__
void add(int n, float *x, float *y, float *z)
{


    int index = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;

    int stride = blockDim.x * blockDim.y * blockDim.z * gridDim.x;

    for (int i = index; i < n; i += stride){
        if(index == 0){
         printf("0 0 is running i=%d\n", i);
        }
        z[i] = x[i] + y[i];
    }
}

int main()
{
    int n = 32768;
    int blockSize = 8;
    dim3 sizevec(32, blockSize, blockSize);
    float *x, *y, *z;
    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&y, n*sizeof(float));
    hipMallocManaged(&z, n*sizeof(float));

    //init
    printf("Init...\n");
    for(int i = 0; i < n; i++){
        x[i] = i % 8;
        y[i] = i % 25;
        z[i] = 0;
    }
    printf("Init complete\n");


    printf("Calc...\n");
    clock_t begin = clock();
    add<<<1, sizevec>>>(n, x, y, z);
    hipDeviceSynchronize();
    printf("Calc complete\n");
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("Time spent: %f\n", time_spent);

    for(int i = 0; i < n; i++){
    if(z[i] != x[i] + y[i]){
       printf("ERROR: Expected %d, got %d\n", x[i]+y[i], z[i]);
    }
    }
    printf("\n");

    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}
