
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>




__global__
void add(int n, float *x, float *y, float *z)
{
    int index = blockIdx.x * blockDim.x * blockDim.y * blockDim.z + threadIdx.x;
    int stride = blockDim.x * blockDim.y * blockDim.z * gridDim.x;

    for (int i = index; i < n; i += stride){
        z[i] = x[i] + y[i];
    }
}

int main()
{
    int n = 32768;
    int blockSize = 8;
    dim3 sizevec(blockSize, blockSize, blockSize);
    float *x, *y, *z;
    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&y, n*sizeof(float));
    hipMallocManaged(&z, n*sizeof(float));

    //init
    printf("Init...\n");
    for(int i = 0; i < n; i++){
        x[i] = i % 8;
        y[i] = i % 25;
        z[i] = 0;
    }
    printf("Init complete\n");


    printf("Calc...\n");
    clock_t begin = clock();
    add<<<1, sizevec>>>(n, x, y, z);
    hipDeviceSynchronize();
    printf("Calc complete\n");
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("Time spent: %f\n", time_spent);

    for(int i = 0; i < n; i++){
    if(z[i] != x[i] + y[i]){
       printf("ERROR: Expected %d, got %d\n", x[i]+y[i], z[i]);
    }
    }
    printf("\n");

    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}
