
#include <hip/hip_runtime.h>
#include <stdio.h>




__global__
void add(int n, float *x, float *y, float *z)
{
    printf("Calc...\n");
    for(int i = 0; i < n; i++){
        z[i] = x[i] + y[i];
    }
    printf("Calc complete\n");
}

int main()
{
    int n = 1000000000;
    float *x, *y, *z;
    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&y, n*sizeof(float));
    hipMallocManaged(&z, n*sizeof(float));

    //init
    printf("Init...\n");
    for(int i = 0; i < n; i++){
        x[i] = (i * 2) % 13;
        y[i] = (i * 3) % 25;
        z[i] = 0;
    }
    printf("Init complete\n");

    add<<<1, 256>>>(n, x, y, z);

    hipDeviceSynchronize();

    for(int i = 0; i < n; i++){
        if(z[i] != x[i] + y[i]){
            printf("ERROR: Expected %d, got %d\n", x[i]+y[i], z[i]);
        }
    }
    printf("\n");

    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}
