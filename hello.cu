
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>




__global__
void add(int n, float *x, float *y, float *z)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride){
        z[i] = x[i] + y[i];
    }
}

int main()
{
    int n = 100000000;
    float *x, *y, *z;
    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&y, n*sizeof(float));
    hipMallocManaged(&z, n*sizeof(float));

    //init
    printf("Init...\n");
    for(int i = 0; i < n; i++){
        x[i] = i % 8;
        y[i] = i % 25;
        z[i] = 0;
    }
    printf("Init complete\n");

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    printf("Calc...\n");
    clock_t begin = clock();
    add<<<256, 256>>>(n, x, y, z);
    hipDeviceSynchronize();
    printf("Calc complete\n");
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("Time spent: %f\n", time_spent);

    for(int i = 0; i < n; i++){
    if(z[i] != x[i] + y[i]){
        printf("ERROR: Expected %d, got %d\n", x[i]+y[i], z[i]);
    }
    }
    printf("\n");

    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}
