
#include <hip/hip_runtime.h>
#include <stdio.h>




__global__
void add(int n, float *x, float *y, float *z)
{
    for(int i = 0; i < n; i++){
        z[i] = x[i] + y[i];
    }
}

int main()
{
    int n = 1000000;
    float *x, *y, *z;
    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&y, n*sizeof(float));
    hipMallocManaged(&z, n*sizeof(float));

    //init
    for(int i = 0; i < n; i++){
        x[i] = (i * 2) % 13;
        y[i] = (i * 3) % 25;
        z[i] = 0;
    }

    add<<<1, 1>>>(n, x, y, z);

    for(int i = 0; i < n; i++){
        if(z[i] != x[i] + y[i]){
            printf("ERROR: Expected %d, got %d", x[i]+y[i], z[i]);
        }
    }

    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}
