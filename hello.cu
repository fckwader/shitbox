
#include <hip/hip_runtime.h>
#include <stdio.h>




__global__
void add(int n, float *x, float *y, float *z)
{

    for(int i = 0; i < n; i++){
        z[i] = x[i] + y[i];
    }

}

int main()
{
    int n = 1000000000;
    float *x, *y, *z;
    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&y, n*sizeof(float));
    hipMallocManaged(&z, n*sizeof(float));

    //init
    printf("Init...\n");
    for(int i = 0; i < n; i++){
        x[i] = (i * 2) % 13;
        y[i] = (i * 3) % 25;
        z[i] = 0;
    }
    printf("Init complete\n");

    printf("Calc...\n");
    add<<<1, 256>>>(n, x, y, z);
    hipDeviceSynchronize();
    printf("Calc complete\n");

    for(int i = 0; i < n; i++){
        if(z[i] != x[i] + y[i]){
            printf("ERROR: Expected %d, got %d\n", x[i]+y[i], z[i]);
        }
    }
    printf("\n");

    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}
