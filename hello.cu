
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>




__global__
void add(int n, float *x, float *y, float *z)
{
    printf("X %d, Y %d, Z %d\n", blockDim.x, blockDim.y, blockDim.z);
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride){
        if(index == 0){
         //   printf("0 0 is running i=%d\n", i);
        }
        z[i] = x[i] + y[i];
    }
}

int main()
{
    int n = 32768;
    int blockSize = 64;
    dim3 sizevec(blockSize, blockSize);
    float *x, *y, *z;
    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&y, n*sizeof(float));
    hipMallocManaged(&z, n*sizeof(float));

    //init
    printf("Init...\n");
    for(int i = 0; i < n; i++){
        x[i] = i % 8;
        y[i] = i % 25;
        z[i] = 0;
    }
    printf("Init complete\n");


    printf("Calc...\n");
    clock_t begin = clock();
    add<<<1, sizevec>>>(n, x, y, z);
    hipDeviceSynchronize();
    printf("Calc complete\n");
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("Time spent: %f\n", time_spent);

    for(int i = 0; i < n; i++){
    if(z[i] != x[i] + y[i]){
       // printf("ERROR: Expected %d, got %d\n", x[i]+y[i], z[i]);
    }
    }
    printf("\n");

    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}
