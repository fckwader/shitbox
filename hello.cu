
#include <hip/hip_runtime.h>
#include <stdio.h>




__global__
void add(int n, float *x, float *y, float *z)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride){
        z[i] = x[i] + y[i];
    }
}

int main()
{
    int n = 1000000;
    float *x, *y, *z;
    hipMallocManaged(&x, n*sizeof(float));
    hipMallocManaged(&y, n*sizeof(float));
    hipMallocManaged(&z, n*sizeof(float));

    //init
    printf("Init...\n");
    for(int i = 0; i < n; i++){
        x[i] = i % 8;
        y[i] = i % 25;
        z[i] = 0;
    }
    printf("Init complete\n");

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    printf("Calc...\n");
    add<<<256, 256>>>(n, x, y, z);
    hipDeviceSynchronize();
    printf("Calc complete\n");

    for(int i = 0; i < n; i++){
    if(z[i] != x[i] + y[i]){
        printf("ERROR: Expected %d, got %d\n", x[i]+y[i], z[i]);
    }
    }
    printf("\n");

    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}
