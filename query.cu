
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {

  int nDevices;
  hipGetDeviceCount(&nDevices);

  printf("Number of devices: %d\n", nDevices);

  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
  }
}